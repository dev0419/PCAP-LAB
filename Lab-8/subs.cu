#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void replaceRows(int m, int n, int* matrix) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        for (int col = 0; col < n; col++) {
            matrix[row * n + col] = pow(matrix[row * n + col], row + 1);
        }
    }
}

int main() {
    int m, n;
    printf("Enter the dimensions of the matrix (MxN): ");
    scanf("%d %d", &m, &n);

    // Allocate and initialize the matrix on the host
    int* h_matrix = (int*)malloc(m * n * sizeof(int));
    printf("Enter the elements of the matrix:\n");
    for (int i = 0; i < m * n; i++) {
        scanf("%d", &h_matrix[i]);
    }

    // Allocate memory on the GPU
    int* d_matrix;
    hipMalloc((void**)&d_matrix, m * n * sizeof(int));

    // Copy the matrix from host to device
    hipMemcpy(d_matrix, h_matrix, m * n * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;

    // Launch the kernel to replace rows
    replaceRows<<<numBlocks, blockSize>>>(m, n, d_matrix);

    // Copy the modified matrix back to the host
    hipMemcpy(h_matrix, d_matrix, m * n * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix);

    // Print the modified matrix
    printf("Modified Matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", h_matrix[i * n + j]);
        }
        printf("\n");
    }

    // Free host memory
    free(h_matrix);

    return 0;
}
