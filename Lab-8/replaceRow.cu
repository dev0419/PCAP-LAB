#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
__global__ void replace(int m,int n,int* mat){
    int row = threadIdx.x + blockDim.x*blockIdx.x;
    if(row < m){
        for(int col = 0;col < n;col++){
            mat[row*n+col] = pow(mat[row*n + col], row + 1);
        }
    }
}

int main(){
    int m,n;
    printf("enter the dimensions:\n");
    scanf("%d %d",&m,&n);
    int* mat = (int*)malloc(sizeof(int)*m*n);
    printf("Enter the elements:\n");
    for(int i = 0;i < m*n;i++)
      scanf("%d",&mat[i]);
    int* d_mat;
    hipMalloc((void**)&d_mat,sizeof(int)*m*n);
    hipMemcpy(d_mat,mat,sizeof(int)*m*n,hipMemcpyHostToDevice);
    replace<<<1,m>>>(m,n,d_mat);
    hipMemcpy(mat,d_mat,sizeof(int)*m*n,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0; i < m;i++){
        for(int j = 0;j < n;j++){
            printf("%d ", mat[i*n+j]);
        }
        printf("\n");
    }
    hipFree(d_mat);
    free(mat);
}
