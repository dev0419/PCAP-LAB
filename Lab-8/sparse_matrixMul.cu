#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void csr(int num_rows,int* data,int* col_index,int* row_ptr,int* x,int* y){
    int row = blockDim.x*blockIdx.x + threadIdx.x;
    if(row < num_rows){
        int res = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row+1];
        for(int i = row_start;i < row_end;i++){
            res += data[i]*x[col_index[i]]; 
        }
        y[row] = res;
    }
}

void csr(int n,int m,int non_zero_count,int* data,int* col_index,int* row_ptr,int* x, int* y){
    int* d_data,*d_col_index,*d_row_ptr,*d_x,*d_y;
    hipMalloc((void**)&d_data,sizeof(int)*non_zero_count);
    hipMalloc((void**)&d_x,sizeof(int)*m);
    hipMalloc((void**)&d_y,sizeof(int)*n);
    hipMalloc((void**)&d_row_ptr,sizeof(int)*(n+1));
    hipMalloc((void**)&d_col_index,sizeof(int)*non_zero_count);
    hipMemcpy(d_data,data,sizeof(int)*non_zero_count,hipMemcpyHostToDevice);
    hipMemcpy(d_x,x,sizeof(int)*m,hipMemcpyHostToDevice);
    hipMemcpy(d_col_index,col_index,sizeof(int)*non_zero_count,hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr,row_ptr,sizeof(int)*(n + 1),hipMemcpyHostToDevice);
    csr<<<1,n>>>(n,d_data,d_col_index,d_row_ptr,d_x,d_y);
    hipMemcpy(y,d_y,sizeof(int)*m,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0; i < m;i++)
      printf("%d ",y[i]);
    printf("\n");
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_data);
    hipFree(d_row_ptr);
    hipFree(d_col_index);
}

int main(){
    int m,n,non_zero_count=0;
    printf("Enter the dimensions:\n");
    scanf("%d %d", &n,&m);
    int* mat = (int*)malloc(sizeof(int)*n*m);
    printf("Enter a sparse matrix:\n");
    for(int i = 0;i < n;i++){
        for(int j = 0; j < m; j++){
            int k = i*n+j;
            scanf("%d",&mat[k]);
            if(mat[k] != 0){
                non_zero_count += 1; 
            }
        }
    }
    int* x = (int*)malloc(sizeof(int)*m);
    int* y = (int*)malloc(sizeof(int)*n);
    int* row_ptr = (int*)calloc((m+1),sizeof(int));
    int* col_index = (int*)malloc(sizeof(int)*non_zero_count);
    int* data = (int*)malloc(sizeof(int)*non_zero_count); 
    printf("Enter the column vector:\n");
    for(int i = 0; i < m;i++){
        scanf("%d",&x[i]);
    }
    int id = 0; 
    for(int i = 0;i < n;i++){
        for(int j = 0;j < m;j++){
            int k = i*n + j;
            if(mat[k] != 0){
                data[id] = mat[k];
                col_index[id] = j;
                id += 1;
            }
            row_ptr[i + 1] = id; 
        }   
    }
    csr(n,m,non_zero_count,data,col_index,row_ptr,x,y);
    return 0;
}
