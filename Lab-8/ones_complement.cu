#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void complement(int* a,int* b,int m,int n){
    int i = threadIdx.y + blockDim.y*blockIdx.y;
    int j = threadIdx.x + blockDim.x*blockIdx.x;
    int num,binary_num=0,base=1,mask;
    if(i < m && j < n){
        if(i == 0||i == m - 1||j == 0||j == n - 1){
            b[i*n+j] = a[i*n+j];
        } else{
            num = a[i*n + j];
            binary_num = 0;
            mask = ~0; 
            while(num & mask){
                mask <<= 1;
            }
            num = ~num & ~mask;
            while(num > 0){
                binary_num += (num % 2) * base;
                num /= 2;
                base *= 10;
            }
            b[i*n+j] = binary_num;
        }
    }
}

int main(){
    int* a,*b,m,n;
    printf("Enter m,n:\n");
    scanf("%d %d",&m,&n);
    a = (int*)malloc(sizeof(int)*m*n);
    b = (int*)malloc(sizeof(int)*m*n);
    printf("Enter the matrix:\n");
    for(int i = 0;i < m;i ++){
        for(int j = 0;j < n;j++){
            scanf("%d",&a[i*n+j]);
        }
    }
    int* d_a,*d_b;
    hipMalloc((void**)&d_a,sizeof(int)*m*n);
    hipMalloc((void**)&d_b,sizeof(int)*m*n);
    hipMemcpy(d_a,a,sizeof(int)*m*n,hipMemcpyHostToDevice);
    dim3 block_size(16,16);
    dim3 num_blocks((n + block_size.x - 1)/block_size.x,(m + block_size.y - 1)/block_size.y);
    complement<<<num_blocks,block_size>>>(d_a,d_b,m,n);
    hipMemcpy(b,d_b,sizeof(int)*m*n,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            printf("%d ",b[i*n + j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);
}
