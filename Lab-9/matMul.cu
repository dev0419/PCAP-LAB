#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2
#define WIDTH 4

__device__ int getTid(){
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int rowInBlock = threadIdx.y;
    int colInBlock = threadIdx.x;
    int globalRow = blockRow * blockDim.y + rowInBlock;
    int globalCol = blockCol * blockDim.x + colInBlock;
    return (globalRow*WIDTH + globalCol);
}

__global__ void matMul(int* a,int* b, int* c){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int sum = 0;
    for (int k = 0; k < WIDTH; k++)
        sum += a[row*WIDTH + k] * b[k*WIDTH + col];
    c[row*WIDTH + col] = sum;
} 

int main(){
    int* matA,*matB,*matC,*da,*db,*dc;
    printf("Enter the elements of (4x4) matrix A:\n");
    matA = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    for (int i = 0; i < WIDTH*WIDTH; i++)
        scanf("%d",&matA[i]);
    printf("Enter the elements of (4x4) matrix B:\n");
    matB = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    for (int i = 0; i < WIDTH*WIDTH; i++)
        scanf("%d",&matB[i]);
    matC = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**) &da,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**) &db,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**) &dc,sizeof(int)*WIDTH*WIDTH);
    hipMemcpy(da,matA,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(db,matB,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH,TILE_WIDTH);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(matC,dc,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < WIDTH; i++){
        for (int j = 0; j < WIDTH; j++){
            printf("%6d ",matC[i*WIDTH + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}
