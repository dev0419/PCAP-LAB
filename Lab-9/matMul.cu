#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define WIDTH 4
#define TILE_WIDTH 2

__global__ void matMul(int* a, int* b, int* c) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the thread is within the matrix dimensions
    if (row < WIDTH && col < WIDTH) {
        int sum = 0;
        for (int k = 0; k < WIDTH; k++) {
            sum += a[row * WIDTH + k] * b[k * WIDTH + col];
        }
        c[row * WIDTH + col] = sum;
    }
}

int main() {
    int* matA, *matB, *matC;
    int *da, *db, *dc;

    matA = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    printf("Enter the elements of the 4x4 matrix A:\n");
    for (int i = 0; i < WIDTH * WIDTH; i++)
        scanf("%d", &matA[i]);

    matB = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    printf("Enter the elements of the 4x4 matrix B:\n"); // Fixed the prompt
    for (int i = 0; i < WIDTH * WIDTH; i++)
        scanf("%d", &matB[i]);

    matC = (int*)malloc(sizeof(int) * WIDTH * WIDTH);

    hipMalloc((void**)&da, sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void**)&db, sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void**)&dc, sizeof(int) * WIDTH * WIDTH);

    hipMemcpy(da, matA, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);

    dim3 grid_conf(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH, TILE_WIDTH);

    matMul<<<grid_conf, block_conf>>>(da, db, dc);

    hipMemcpy(matC, dc, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%d ", matC[i * WIDTH + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    free(matA);
    free(matB);
    free(matC);

    return 0;
}
