#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void matMul(int* a,int* b,int* c){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int res = 0;
    if(row < WIDTH && col < WIDTH){
        for(int k = 0;k < WIDTH;k++){
            res = a[row*WIDTH + k]*b[k*WIDTH + col];
        }
        c[row*WIDTH + col] = res;
    }
}

int main(){
    int* a,*b,*c,*da,*db,*dc;
    a = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    b = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    c = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    printf("Enter 4x4 matrix a:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&a[i]);
    printf("Enter 4x4 matrix b:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&b[i]);
    hipMalloc((void**)&da,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&db,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&dc,sizeof(int)*WIDTH*WIDTH);
    hipMemcpy(da,a,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(db,b,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH,TILE_WIDTH);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(c,dc,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0; i < WIDTH;i++){
        for(int j = 0; j < WIDTH;j++){
            printf("%d ",c[i*WIDTH + j]);       
        }
        printf("\n");
    }
    return 0;
}
