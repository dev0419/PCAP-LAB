#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2 
#define WIDTH 4
__global__ void matMul(int* a,int* b,int* c){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int pval = 0;
    if(row < WIDTH && col < WIDTH){
        for(int k = 0;k < WIDTH;k++)
            pval += a[row*WIDTH + k]*b[k*WIDTH + col];
        c[row*WIDTH + col] = pval;
    }
}

void perform_matMul(int* a,int* b,int* c){
    int* da,*db,*dc,size;
    size = sizeof(int)*WIDTH*WIDTH;
    hipMalloc((void**)&da,size);
    hipMalloc((void**)&db,size);
    hipMalloc((void**)&dc,size);
    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(WIDTH,WIDTH);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

int main(){
    int* a,*b,*c;
    a = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    b = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    c = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    printf("Enter 4x4 matrix A:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&a[i]);
    printf("Enter 4x4 matrix B:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&b[i]);
    perform_matMul(a,b,c);
    printf("Result:\n");
    for(int i = 0;i < WIDTH;i++){
        for(int j = 0;j < WIDTH;j++){
            printf("%d ",c[i*WIDTH + j]);
        }
        printf("\n");
    }
    free(a);
    free(b);
    free(c);
    return 0;
}
