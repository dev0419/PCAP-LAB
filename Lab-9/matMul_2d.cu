#include "hip/hip_runtime.h"
%%cuda --name prg1.cu 
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2
#define WIDTH 4
__global__ void matMul(int* a,int* b,int* c){
    int row = threadIdx.y + blockDim.y*blockIdx.y;
    int col = threadIdx.x + blockDim.x*blockIdx.x;
    if(row < WIDTH && col < WIDTH){
        int sum = 0;
        for(int k = 0;k < WIDTH;k++){
            sum += a[row*WIDTH + k]*b[k*WIDTH + col];
        }
        c[row*WIDTH + col] = sum;
    }
}

int main(){
    int* matA,*matB,*matC,*da,*db,*dc;
    matA = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    matB = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    matC = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    printf("Enter the elements of matrix of A (4x4):\n");
    for(int i = 0;i < WIDTH;i++){
        for(int j = 0;j < WIDTH;j++){
            scanf("%d",&matA[i*WIDTH + j]);
        }
    }
    printf("Enter the elements of matrix of B (4x4):\n");
    for(int i = 0;i < WIDTH;i++){
        for(int j = 0;j < WIDTH;j++){
            scanf("%d",&matB[i*WIDTH + j]);
        }
    }
    hipMalloc((void**)&da,WIDTH*WIDTH*sizeof(int));
    hipMalloc((void**)&db,WIDTH*WIDTH*sizeof(int));
    hipMalloc((void**)&dc,WIDTH*WIDTH*sizeof(int));
    hipMemcpy(da,matA,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(db,matB,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH,TILE_WIDTH);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(matC,dc,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < WIDTH;i++){
        for(int j = 0;j < WIDTH;j++){
            printf("%d ",matC[i*WIDTH+j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(matA);
    free(matB);
    free(matC);
    return 0;
}
