#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 2
#define WIDTH 4
#define MASK_WIDTH 3

__global__ void convolution(int *input, int *mask, int *output) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    for (int i = 0; i < MASK_WIDTH; i++) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            int inputRow = row + i - MASK_WIDTH / 2;
            int inputCol = col + j - MASK_WIDTH / 2;

            if (inputRow >= 0 && inputRow < WIDTH && inputCol >= 0 && inputCol < WIDTH) {
                sum += input[inputRow * WIDTH + inputCol] * mask[i * MASK_WIDTH + j];
            }
        }
    }

    output[row * WIDTH + col] = sum;
}

int main() {
    int *input, *mask, *output, *d_input, *d_mask, *d_output;

    printf("Enter the elements of (4x4) input matrix:\n");
    input = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    for (int i = 0; i < WIDTH * WIDTH; i++) {
        scanf("%d", &input[i]);
    }

    printf("Enter the elements of (3x3) mask matrix:\n");
    mask = (int*)malloc(sizeof(int) * MASK_WIDTH * MASK_WIDTH);
    for (int i = 0; i < MASK_WIDTH * MASK_WIDTH; i++) {
        scanf("%d", &mask[i]);
    }

    output = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void**)&d_input, sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void**)&d_mask, sizeof(int) * MASK_WIDTH * MASK_WIDTH);
    hipMalloc((void**)&d_output, sizeof(int) * WIDTH * WIDTH);

    hipMemcpy(d_input, input, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, sizeof(int) * MASK_WIDTH * MASK_WIDTH, hipMemcpyHostToDevice);

    dim3 grid_conf(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH, TILE_WIDTH);

    convolution<<<grid_conf, block_conf>>>(d_input, d_mask, d_output);

    hipMemcpy(output, d_output, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

    printf("Result of Convolution:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%6d ", output[i * WIDTH + j]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
    free(input);
    free(mask);
    free(output);

    return 0;
}
