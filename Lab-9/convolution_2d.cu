#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2 
#define WIDTH 4
#define MASK_WIDTH 3

__global__ void convolution(int* input, int* mask, int* output) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;
    for (int i = 0; i < MASK_WIDTH; i++) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            int inputRow = row + i - MASK_WIDTH / 2;
            int inputCol = col + j - MASK_WIDTH / 2;
            if (inputRow >= 0 && inputRow < WIDTH && inputCol >= 0 && inputCol < WIDTH) {
                sum += input[inputRow * WIDTH + inputCol] * mask[i * MASK_WIDTH + j];
            }
        }
    }
    output[row * WIDTH + col] = sum;
}

int main() {
    int *input, *mask, *output, *d_input, *d_mask, *d_output;
    input = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    output = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
    mask = (int*)malloc(sizeof(int) * MASK_WIDTH * MASK_WIDTH);
    printf("Enter the (4x4) input matrix:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            scanf("%d", &input[i * WIDTH + j]);
        }
    }
    printf("Enter the (3x3) mask matrix:\n");
    for (int i = 0; i < MASK_WIDTH; i++) {
        for (int j = 0; j < MASK_WIDTH; j++) {
            scanf("%d", &mask[i * MASK_WIDTH + j]);
        }
    }
    hipMalloc((void**)&d_input, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void**)&d_mask, MASK_WIDTH * MASK_WIDTH * sizeof(int));
    hipMalloc((void**)&d_output, WIDTH * WIDTH * sizeof(int));
    hipMemcpy(d_input, input, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, MASK_WIDTH * MASK_WIDTH * sizeof(int), hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH, TILE_WIDTH);
    convolution<<<grid_conf, block_conf>>>(d_input, d_mask, d_output);
    hipMemcpy(output, d_output, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);
    printf("After performing convolution:\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%d ", output[i * WIDTH + j]);
        }
        printf("\n");
    }
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
    free(input);
    free(output);
    free(mask);
}
