
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void addRow(int* a,int* b,int* c,int wa,int wb){
    int ridA = threadIdx.x;
    for (int cidB = 0; cidB < wb; cidB++)
        c[ridA*wb + cidB] = a[ridA*wb + cidB] + b[ridA*wb + cidB];
}

__global__ void addCol(int* a,int* b,int* c, int ha,int hb){
    int cidB = threadIdx.x;
    for (int ridA = 0; ridA < ha; ridA++)
        c[ridA*hb + cidB] = a[ridA*hb + cidB] + b[ridA*hb + cidB]; 
}

__global__ void addElement(int* a, int* b, int* c,int wa,int wb){
    int row = threadIdx.y;
    int col = threadIdx.x;
    c[row*wb + col] = (a[row*wb + col] + b[row*wb + col]);
}

void printMatrix(int* a, int row, int col){
    for (int i = 0; i < row; i++){
        for (int j = 0; j < col; j++){
            printf("%d ",a[i*col + j]);
        }
        printf("\n");
    }
}

void addMat(int* a,int* b,int* c,int wa,int wb,int ha, int hb){
    int* d_A,*d_B,*d_C;
    int size = ha*wb*sizeof(int);
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    hipMemcpy(d_A,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,b,size,hipMemcpyHostToDevice);
    addRow<<<1,ha>>>(d_A,d_B,d_C,wa,wb);
    hipMemcpy(c,d_C,size,hipMemcpyDeviceToHost);
    printf("Result after row-wise addition:\n");
    printMatrix(c,ha,wb);
    addCol<<<1,wb>>>(d_A,d_B,d_C,ha,hb);
    hipMemcpy(c,d_C,size,hipMemcpyDeviceToHost);
    printf("Result after column-wise addition:\n");
    printMatrix(c,ha,wb);
    addElement<<<1,wb>>>(d_A,d_B,d_C,wa,wb);
    printf("Result after element-wise addition:\n");
    printMatrix(c,ha,wb);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(){
    int* a,*b,*c,m1,n1,m2,n2;
    printf("Enter the rows and columns of matrix a:\n");
    scanf("%d %d",&m1,&n1);
    a = (int*)malloc(m1*n1*sizeof(int));
    printf("Enter the elements of matrix A:\n");
    for (int i = 0; i < m1; i++)
        for (int j = 0; j < n1; j++)
            scanf("%d",&a[i*n1 + j]);
    printf("Enter the rows and columns of matrix b:\n");
    scanf("%d %d",&m2,&n2);
    b = (int*)malloc(m2*n2*sizeof(int));
    printf("Enter the elements of matrix b:\n");
    for (int i = 0; i < m2; i++)
        for (int j = 0; j < n2; j++)
            scanf("%d",&b[i*n2 + j]);
    c = (int*)malloc(m1*n2*sizeof(int));
    addMat(a,b,c,n1,n2,m1,m2);
    free(a);
    free(b);
    free(c);
    return 0;
}

