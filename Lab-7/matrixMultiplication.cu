#include "hip/hip_runtime.h"
%%cuda --name prg3.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Define the kernel to perform matrix multiplication row-wise
__global__ void multiply_rowWise(int* a, int* b, int* c, int wa, int wb) {
    int ridA = threadIdx.x;
    int sum;

    for (int cidB = 0; cidB < wb; cidB++) {
        sum = 0;

        for (int k = 0; k < wa; k++) {
            sum += (a[ridA * wa + k] * b[k * wb + cidB]);
        }

        c[ridA * wb + cidB] = sum;
    }
}

// Define the kernel to perform matrix multiplication column-wise
__global__ void multiply_Colwise(int* a, int* b, int* c, int ha, int wa) {
    int cidB = threadIdx.x;
    int wb = blockDim.x; // Corrected variable declaration

    for (int ridA = 0; ridA < ha; ridA++) {
        int sum = 0; // Initialize 'sum' to 0 for each column

        for (int k = 0; k < wa; k++) {
            sum += (a[ridA * wa + k] * b[k * wb + cidB]); // Perform element-wise multiplication and accumulate the result
        }

        c[ridA * wb + cidB] = sum; // Store the accumulated 'sum' in the corresponding position of matrix 'c'
    }
}

// Define the kernel to perform matrix multiplication
__global__ void multiplyKernel(int* a, int* b, int* c, int wa, int wb) {
    int ridA = threadIdx.y; // Use threadIdx.y for rows
    int cidB = threadIdx.x; // Use threadIdx.x for columns

    int sum = 0;

    for (int k = 0; k < wa; k++) {
        sum += (a[ridA * wa + k] * b[k * wb + cidB]);
    }

    c[ridA * wb + cidB] = sum;
}

// Function to perform matrix multiplication
void Multiply(int* a, int* b, int* c, int wa, int wb, int ha) {
    int* d_A, * d_B, * d_C; // Declare device memory pointers

    // Calculate the required memory size
    int size = ha * wb * sizeof(int);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);

    // Launch the kernel to perform matrix multiplication row-wise
    multiply_rowWise<<<1, ha>>>(d_A, d_B, d_C, wa, wb);

    // Copy the result from device to host
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("Result of row-wise matrix multiplication:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wb; j++) {
            printf("%d ", c[i * wb + j]);
        }
        printf("\n");
    }

    // Launch the kernel to perform matrix multiplication column-wise
    multiply_Colwise<<<1, wb>>>(d_A, d_B, d_C, ha, wa);

    // Copy the result from device to host
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("\nResult of column-wise matrix multiplication:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wb; j++) {
            printf("%d ", c[i * wb + j]);
        }
        printf("\n");
    }

    // Launch the kernel to perform matrix multiplication
    dim3 blockSize(wb, ha);
    multiplyKernel<<<1, blockSize>>>(d_A, d_B, d_C, wa, wb);

    // Copy the result from device to host
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("\nResult of matrix multiplication element-wise:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wb; j++) {
            printf("%d ", c[i * wb + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int* a, * b, * c;
    int wa, ha, wb, hb;

    printf("Enter the dimensions of matrix A (ha wa):\n");
    scanf("%d %d", &ha, &wa);
    printf("Enter the dimensions of matrix B (hb wb):\n");
    scanf("%d %d", &hb, &wb);

    // Check if matrix dimensions are compatible for multiplication
    if (wa != hb) {
        printf("Matrix dimensions are not compatible for multiplication.\n");
        return 1;
    }

    // Allocate memory for matrices a, b, and c
    a = (int*)malloc(ha * wa * sizeof(int));
    b = (int*)malloc(hb * wb * sizeof(int));
    c = (int*)malloc(ha * wb * sizeof(int));

    printf("Enter the elements of matrix A:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wa; j++) {
            scanf("%d", &a[i * wa + j]);
        }
    }

    printf("Enter the elements of matrix B:\n");
    for (int i = 0; i < hb; i++) {
        for (int j = 0; j < wb; j++) {
            scanf("%d", &b[i * wb + j]);
        }
    }

    // Call the Multiply function to perform matrix multiplication
    Multiply(a, b, c, wa, wb, ha);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
