#include "hip/hip_runtime.h"
%%cuda --name prg8.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void multiply_rowWise(int* a, int* b, int* c, int wa, int wb) {
    int ridA = threadIdx.x;
    int sum;
    for (int cidB = 0; cidB < wb; cidB++) {
        sum = 0;
        for (int k = 0; k < wa; k++) 
            sum += (a[ridA * wa + k] * b[k * wb + cidB]);
        c[ridA * wb + cidB] = sum;
    }
}

__global__ void multiply_Colwise(int* a, int* b, int* c, int ha, int wa) {
    int cidB = threadIdx.x;
    int wb = blockDim.x;

    for (int ridA = 0; ridA < ha; ridA++) {
        int sum = 0; 
        for (int k = 0; k < wa; k++) 
            sum += (a[ridA * wa + k] * b[k * wb + cidB]); 
        c[ridA * wb + cidB] = sum;
    }
}

__global__ void multiplyKernel(int* a, int* b, int* c, int wa, int wb) {
    int ridA = threadIdx.y; 
    int cidB = threadIdx.x; 
    int sum = 0;
    for (int k = 0; k < wa; k++) 
        sum += (a[ridA * wa + k] * b[k * wb + cidB]);
    c[ridA * wb + cidB] = sum;
}

void printMatrix(int* a,int row,int col){
    for(int i = 0;i < row;i++){
        for(int j = 0;j < col;j++){
          printf("%d ", a[i*col + j]);
        }
        printf("\n");
    }
}


void Multiply(int* a, int* b, int* c, int wa, int wb, int ha) {
    int* d_A, * d_B, * d_C; 
    int size = ha * wb * sizeof(int);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);

    multiply_rowWise<<<1, ha>>>(d_A, d_B, d_C, wa, wb);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("Result of row-wise matrix multiplication:\n");
    printMatrix(c,ha,wb);

    multiply_Colwise<<<1, wb>>>(d_A, d_B, d_C, ha, wa);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
    printf("\nResult of column-wise matrix multiplication:\n");
    printMatrix(c,ha,wb);

    dim3 blockSize(wb, ha);
    multiplyKernel<<<1, blockSize>>>(d_A, d_B, d_C, wa, wb);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    printf("\nResult of matrix multiplication element-wise:\n");
    printMatrix(c,ha,wb);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int* a, * b, * c;
    int wa, ha, wb, hb;

    printf("Enter the dimensions of matrix A (ha wa):\n");
    scanf("%d %d", &ha, &wa);
    printf("Enter the dimensions of matrix B (hb wb):\n");
    scanf("%d %d", &hb, &wb);
    
    a = (int*)malloc(ha * wa * sizeof(int));
    b = (int*)malloc(hb * wb * sizeof(int));
    c = (int*)malloc(ha * wb * sizeof(int));

    printf("Enter the elements of matrix A:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wa; j++) {
            scanf("%d", &a[i * wa + j]);
        }
    }

    printf("Enter the elements of matrix B:\n");
    for (int i = 0; i < hb; i++) {
        for (int j = 0; j < wb; j++) {
            scanf("%d", &b[i * wb + j]);
        }
    }

    Multiply(a, b, c, wa, wb, ha);
    free(a);
    free(b);
    free(c);
    return 0;
}
