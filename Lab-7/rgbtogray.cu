
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void rgbtogray(unsigned char* grayImg,unsigned char* rgbImg,int height,int width){
    int col = threadIdx.x + blockDim.x*blockIdx.x;
    int row = threadIdx.y + blockDim.y*blockIdx.y;
    if (row < height && col < width){
        int grayOffset = row*width + col;
        int rbgOffset = grayOffset*3;
        unsigned char r = rgbImg[rbgOffset];
        unsigned char g = rgbImg[rbgOffset + 1];
        unsigned char b = rgbImg[rbgOffset + 2];
        grayImg[grayOffset] = 0.21f * r + 0.72f * g + 0.07f * b;
    }
}

void RgbtoGray(unsigned char* grayImg,unsigned char* rgbImg,int height,int width){
    int rgb_size = height*width*3*sizeof(unsigned char);
    int gray_size = height*width*sizeof(unsigned char);
    unsigned char* d_rgb, *d_gray;
    for (int i = 0; i < width*height*3; i++)
        rgbImg[i] = (i % 256);
    
    printf("Original values:\n");
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int idx = i*width + j;
            printf("(%u %u %u) ", rgbImg[idx], rgbImg[idx + 1],rgbImg[idx + 2]);   
        }
        printf("\n");
    }

    hipMalloc((void**)&d_gray,gray_size);
    hipMalloc((void**)&d_rgb,rgb_size);
    hipMemcpy(d_rgb,rgbImg,rgb_size,hipMemcpyHostToDevice);
    
    dim3 blockSize(32,32);
    dim3 gridSize((width + blockSize.x - 1)/blockSize.x,(height + blockSize.y - 1)/blockSize.y);
    rgbtogray<<<gridSize,blockSize>>>(d_gray,d_rgb,height,width);
    hipMemcpy(grayImg,d_gray,gray_size,hipMemcpyDeviceToHost);
    printf("Converted grayscale values:\n");
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int idx = i*width + j;
            printf("%u ",grayImg[idx]);
        }
        printf("\n");
    }
}

int main(){
    int height = 3, width = 3;
    unsigned char* gray = (unsigned char*)malloc(height*width*sizeof(unsigned char));
    unsigned char* rgb = (unsigned char*)malloc(height*width*3*sizeof(unsigned char));
    RgbtoGray(gray,rgb,height,width);
    return 0;
}
