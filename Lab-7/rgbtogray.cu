#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>
#include<stdlib.h>
#include<string.h>
__global__ void rgbtogray(unsigned char* grayImg,unsigned char* rgbImg, int width, int height){
    int col = threadIdx.x + blockDim.x*blockIdx.x;
    int row = threadIdx.y + blockDim.y*blockIdx.y;
    if (row < height && col < width){
        int grayOffset =  row*width + col;
        int rgbOffset = grayOffset*3;
        unsigned char r = rgbImg[rgbOffset];
        unsigned char g = rgbImg[rgbOffset + 1];
        unsigned char b = rgbImg[rgbOffset + 2];
        grayImg[grayOffset] = 0.21f*r + 0.72f*g + 0.07f*b; 
    }
}

int main() {
    const int width = 3; 
    const int height = 3;
    unsigned char* rgb = (unsigned char*)malloc(3*height*width*sizeof(unsigned char));
    unsigned char* gray = (unsigned char*)malloc(height*width*sizeof(unsigned char));
    unsigned char* d_rgb,*d_gray;
    for(int i = 0; i < 3*width*height;i++)
        rgb[i] = (i%256);
    
    printf("Original RGB values:\n");
    for(int i = 0; i < height;i++){
        for(int j = 0; j < width; j++){
            int idx = 3*(i*width +j);
            printf("(%u, %u, %u) ",rgb[idx],rgb[idx+1],rgb[idx+2]);
        }
        printf("\n");
    }

    hipMalloc((void**)&d_rgb,3*width*height*sizeof(unsigned char));
    hipMalloc((void**)&d_gray,width*height*sizeof(unsigned char));
    hipMemcpy(d_rgb,rgb,3*width*height*sizeof(unsigned char),hipMemcpyHostToDevice);
    dim3 blockSize(32,32);
    dim3 gridSize(((width + blockSize.x - 1)/blockSize.x),((height + blockSize.y - 1)/blockSize.y));
    rgbtogray<<<gridSize,blockSize>>>(d_gray,d_rgb,width,height);
    hipMemcpy(gray,d_gray,height*width*sizeof(unsigned char),hipMemcpyDeviceToHost);
    printf("converted grayscale values:\n");
    for(int i = 0; i < height;i++){
        for(int j = 0; j < width; j++){
            int idx = i*width + j;
            printf("%u ", gray[idx]);
        }
        printf("\n");
    }
    free(rgb);
    free(gray);
    hipFree(d_gray);
    hipFree(d_rgb);
    return 0;
}
