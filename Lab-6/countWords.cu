#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"

#define n 1024
#define nw 1024

__global__ void countWords(char* a, char* w, int* start_index, int* len_words, unsigned int l_w, unsigned int* d_count) {
    int id = threadIdx.x;
    if (len_words[id] < l_w) {
        return;
    }
    int start = start_index[id];
    for (int i = 0; i < l_w; i++) {
        if (a[start + i] != w[i])
            return;
    }
    atomicAdd(d_count, 1);
}

int main() {
    char a[n], w[n];
    char* d_A, * d_W;
    int start_index[nw], len_words[nw];
    int* d_start_index, * d_len_words;
    int len;
    unsigned int count = 0, * d_count, result;
    
    printf("Enter the string:\n");
    scanf(" %[^\n]s", a);

    printf("Enter the word to be searched:\n");
    scanf(" %[^\n]s", w);
    len = strlen(a);
    int i = 0, k = 0;
    
    while (i < len) {
        while (i < len && a[i] == ' ')
            i++;
        start_index[k] = i;
        while (i < len && a[i] != ' ')
            i++;
        len_words[k] = i - start_index[k];
        k++;
    }

    if (len_words[k - 1] == 0)
        k--;
    
    hipMalloc((void**)&d_A, strlen(a) * sizeof(char));
    hipMalloc((void**)&d_W, strlen(w) * sizeof(char));
    hipMalloc((void**)&d_start_index, k * sizeof(int));
    hipMalloc((void**)&d_len_words, k * sizeof(int));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    
    hipMemcpy(d_A, a, strlen(a) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_W, w, strlen(w) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_start_index, start_index, k * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len_words, len_words, k * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    countWords<<<1, k>>>(d_A, d_W, d_start_index, d_len_words, strlen(w), d_count);

    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    printf("Total occurrences of %s: %u\n", w, result);
    
    hipFree(d_A); hipFree(d_W); hipFree(d_start_index); hipFree(d_len_words); hipFree(d_count);

    return 0;
}
