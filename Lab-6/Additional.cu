#include <stdio.h>
#include <hip/hip_runtime.h>
// Scenario (a): Every string is produced parallely
__global__ void copyStr_a(char* str, char* rstr, int len, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < len; i++) {
        rstr[tid * len + i] = str[i];
    }
}

// Scenario (b): Every thread will generate the same character n no of times
__global__ void copyStr_b(char* str, char* rstr, int len, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int pos = tid * len;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < len; j++) {
            rstr[pos + j] = str[j];
        }
        pos += len;
    }
}

int main() {
    char str[] = "hello";
    int n = 3;
    int len = strlen(str);
    char *d_str, *d_rstr;
    char rstr[len * n];

    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_rstr, len * n * sizeof(char));

    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (len * n + blockSize - 1) / blockSize;

    int choice;
    printf("Choose scenario (a or b):\n");
    printf("a) Every string is produced parallely\n");
    printf("b) Every thread will generate the same character n no of times\n");
    scanf("%c", &choice);

    if (choice == 'a') {
        copyStr_a<<<numBlocks, blockSize>>>(d_str, d_rstr, len, n);
    } else if (choice == 'b') {
        copyStr_b<<<numBlocks, blockSize>>>(d_str, d_rstr, len, n);
    } else {
        printf("Invalid choice\n");
        return 1;
    }

    hipMemcpy(rstr, d_rstr, len * n * sizeof(char), hipMemcpyDeviceToHost);

    for (int i = 0; i < len * n; i++) {
        printf("%c", rstr[i]);
    }

    hipFree(d_str);
    hipFree(d_rstr);

    return 0;
}
