#include <stdio.h>
#include <hip/hip_runtime.h>

#include <string.h>

__global__ void createRS(char* d_S, char* d_RS, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int start = (idx * (2 * length - idx + 1)) / 2;

    for (int i = 0; i < length - idx; i++) {
        d_RS[start + i] = d_S[i];
    }
}


int main() {
    char S[100];
    printf("Enter the string: ");
    scanf("%99s", S);

    int length = strlen(S);
    int totalLength = (length * (length + 1)) / 2;  // Sum of the first 'length' natural numbers
    char* RS = (char*)malloc(totalLength + 1);
    RS[totalLength] = '\0';

    char *d_S, *d_RS;

    hipMalloc((void**)&d_S, length * sizeof(char));
    hipMalloc((void**)&d_RS, totalLength * sizeof(char));

    hipMemcpy(d_S, S, length * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = length;
    int gridSize = 1;
    createRS <<<gridSize, blockSize>>> (d_S, d_RS, length);

    hipMemcpy(RS, d_RS, totalLength * sizeof(char), hipMemcpyDeviceToHost);

    printf("Output string RS: %s\n", RS);

    free(RS);
    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}