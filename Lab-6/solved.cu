#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define N 1024

__global__ void charCount(char* a, unsigned int* d_count) {
    int i = threadIdx.x;
    if (a[i] == 'a')
        atomicAdd(d_count, 1);
}

int main() {
    char a[N], *d_A;
    unsigned int count = 0, *d_count, result;
    
    printf("Enter string:\n");
    scanf("%[^\n]s", a); // Changed from "%[^/n]s" to "%[^\n]s" for correct input
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipMalloc((void**)&d_A, strlen(a) * sizeof(char)); // Changed "strlen(A)" to "strlen(a)"
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    hipMemcpy(d_A, a, strlen(a) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error 1: %s\n", hipGetErrorString(err));
    
    charCount<<<1, strlen(a)>>>(d_A, d_count);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); "
    
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost); 
    
    printf("Total occurrences of 'a': %u\n", result);
    printf("Time Taken: %f\n", elapsedTime);
    
    hipFree(d_A);
    hipFree(d_count);
    
    return 0;
}

