#include <stdio.h>
#include <hip/hip_runtime.h>

_global_ void repeatCharacters(const char *A, const int *B, char *output, int elements, int *offsets) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < elements) {
        int repeats = B[idx];
        char character = A[idx];
        int outputPos = offsets[idx];

        for (int j = 0; j < repeats; ++j) {
            output[outputPos + j] = character;
        }
    }
}

int main() {
    int rows, cols;
    char *A;
    int *B;
    char *output;
    int totalSize, totalOutputSize = 0;

    // Ask user for matrix dimensions
    printf("Enter the number of rows and columns for matrices A and B: ");
    scanf("%d %d", &rows, &cols);

    totalSize = rows * cols;

    // Allocate host memory
    A = (char *)malloc(totalSize * sizeof(char));
    B = (int *)malloc(totalSize * sizeof(int));
    int *offsets = (int *)malloc(totalSize * sizeof(int));

    // Initialize host matrices A and B with user input
    printf("Enter the elements of matrix A:\n");
    for (int i = 0; i < totalSize; ++i) {
        scanf(" %c", &A[i]); // Note the space before %c to catch any previous whitespaces
    }

    printf("Enter the elements of matrix B:\n");
    for (int i = 0; i < totalSize; ++i) {
        scanf("%d", &B[i]);
        if (i == 0)
            offsets[i] = 0;
        else
            offsets[i] = offsets[i - 1] + B[i - 1];
        totalOutputSize += B[i];
    }

    // Allocate output string
    output = (char *)malloc((totalOutputSize + 1) * sizeof(char)); // +1 for the null-terminator
    output[totalOutputSize] = '\0'; // Null-terminate the string

    // Allocate device memory
    char *d_A;
    int *d_B, *d_offsets;
    char *d_output;
    hipMalloc((void **)&d_A, totalSize * sizeof(char));
    hipMalloc((void **)&d_B, totalSize * sizeof(int));
    hipMalloc((void **)&d_output, totalOutputSize * sizeof(char));
    hipMalloc((void **)&d_offsets, totalSize * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, totalSize * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, totalSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets, totalSize * sizeof(int), hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((totalSize + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Launch CUDA kernel
    repeatCharacters<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_output, totalSize, d_offsets);

    // Copy output string from device to host
    hipMemcpy(output, d_output, totalOutputSize * sizeof(char), hipMemcpyDeviceToHost);

    // Print the resulting output string
    printf("Output String: %s\n", output);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_output); hipFree(d_offsets);
    free(A); free(B); free(output); free(offsets);

    return 0;
}
