#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void spmv_Csr(int num_rows, float* data, int* col_index, int* row_ptr, float* x, float* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float result = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int ele = row_start; ele < row_end; ele++) {
            result += data[ele] * x[col_index[ele]];
        }
        y[row] = result;
    }
}

int main() {
    int n, m;
    printf("Enter dimensions of matrix: ");
    scanf("%d%d", &n, &m);
    float* h_matrix = (float*)malloc(n * m * sizeof(float));
    printf("Enter elements of matrix:\n");
    for (int i = 0; i < n * m; i++) {
        scanf("%f", h_matrix + i);
    }
    float* h_x = (float*)malloc(m * sizeof(float));
    printf("Enter %d elements of vector x: ", m);
    for (int i = 0; i < m; i++) {
        scanf("%f", h_x + i);
    }
    int* h_row_ptr = (int*)calloc(n + 1, sizeof(int));
    int non_zero_count = 0;
    for (int i = 0; i < n; i++) {
        h_row_ptr[i] = non_zero_count;
        for (int j = 0; j < m; j++) {
            int k = i * m + j;
            if (h_matrix[k] != 0) {
                non_zero_count++;
            }
        }
    }
    h_row_ptr[n] = non_zero_count;
    float* h_data = (float*)malloc(non_zero_count * sizeof(float));
    int* h_col_index = (int*)malloc(non_zero_count * sizeof(int));
    int id = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            int k = i * m + j;
            if (h_matrix[k] != 0) {
                h_data[id] = h_matrix[k];
                h_col_index[id] = j;
                id++;
            }
        }
    }

    float* h_y = (float*)malloc(n * sizeof(float));

    // Declare and allocate device memory
    float* d_data;
    int* d_col_index;
    int* d_row_ptr;
    float* d_x;
    float* d_y;

    hipMalloc((void**)&d_data, non_zero_count * sizeof(float));
    hipMalloc((void**)&d_col_index, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (n + 1) * sizeof(int));
    hipMalloc((void**)&d_x, m * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_data, h_data, non_zero_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, h_col_index, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, m * sizeof(float), hipMemcpyHostToDevice);

    // Set grid and block dimensions
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    // Launch the kernel
    spmv_Csr<<<grid_size, block_size>>>(n, d_data, d_col_index, d_row_ptr, d_x, d_y);

    // Copy the result y from device to host
    hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result vector y
    printf("Result vector y:\n");
    for (int i = 0; i < n; i++) {
        printf("%.2f\n", h_y[i]);
    }

    // Free device memory
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_ptr);
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(h_matrix);
    free(h_x);
    free(h_row_ptr);
    free(h_data);
    free(h_col_index);
    free(h_y);

    return 0;
}
