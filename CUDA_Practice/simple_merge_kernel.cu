
#include <hip/hip_runtime.h>
__global__ void merge_kernel(int* a, int m, int* b, int n, int* c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = m + n;
    int num_threads = blockDim.x * gridDim.x;

    int elements_per_thread = total / num_threads;
    int k_curr = tid * elements_per_thread;
    int k_next;

    if (tid == num_threads - 1) { // Check if this is the last thread
        // Last thread takes all remaining elements
        k_next = total;
    } else {
        // Otherwise, calculate the end index as usual
        k_next = (tid + 1) * elements_per_thread;
    }

    // ...rest of your merge_kernel function...
}
