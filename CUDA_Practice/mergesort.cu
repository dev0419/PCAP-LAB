#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__device__ int co_rank(int k,int* a,int m,int* b,int n){
    int i = min(k,m);
    int j = min(k,n);
    int i_low = max(0,k-m);
    int j_low = max(0,k-n);
    int delta;
    bool flag = true;
    while(flag){
        if(i > 0 && j < n && a[i - 1] > b[j]){
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j += delta;
            i -= delta;
        } else if(j > 0 && i < m &&  b[j] > a[i - 1]){
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            j -= delta;
            i += delta;
        } else{
            flag = false;
        }
    }
    return i;
}

__device__ void merge_sequential(int* a,int m,int* b,int n,int* c){
    int i = 0;
    int j = 0;
    int k = 0;
    while((i < m) && (j < n)){
        if(a[i] < b[j]){
            c[k++] = a[i++];
        } else{
            c[k++] = b[j++];
        }
    } while(i < m){
        c[k++] = a[i++];
    } while(j < n){
        c[k++] = b[j++];
    }
}

__global__ void merge_kernel(int* a,int m,int* b,int n,int* c){
    int tid  = blockIdx.x*blockDim.x + threadIdx.x;
    int total = m + n;
    int k_curr = tid*ceilf((float)total/(gridDim.x*blockDim.x));
    int k_next = min((int)((tid+1)*ceilf((float)total/(gridDim.x*blockDim.x))),total);
    int i_curr = co_rank(k_curr,a,m,b,n);
    int i_next = co_rank(k_next,a,m,b,n);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next; 
    if(tid < total){
        merge_sequential(a + i_curr,i_next - i_curr,b + j_curr,j_curr - j_next,c + k_curr);
    }
}

int main(){
    int m,n,*a,*b,*c,*da,*db,*dc;
    printf("Enter the size m and n:\n");
    scanf("%d %d",&m,&n);
    a = (int*)malloc(sizeof(int)*m);
    b = (int*)malloc(sizeof(int)*n);
    c = (int*)malloc(sizeof(int)*(m + n));
    printf("Enter the sorted array A:\n");
    for(int i = 0;i < m;i++){
        scanf("%d",&a[i]);
    }
    printf("Enter the sorted array B:\n");
    for(int i = 0;i < n;i++){
        scanf("%d",&b[i]);
    }
    hipMalloc((void**)&da,sizeof(int)*m);
    hipMalloc((void**)&db,sizeof(int)*n);
    hipMalloc((void**)&dc,sizeof(int)*(m + n));
    hipMemcpy(da,a,sizeof(int)*m,hipMemcpyHostToDevice);
    hipMemcpy(db,b,sizeof(int)*n,hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = (int)ceil((m + n)/blockSize);
    merge_kernel<<<gridSize,blockSize>>>(da,m,db,n,dc);
    hipMemcpy(c,dc,sizeof(int)*(m + n),hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < (m + n);i++){
        printf("%d ",c[i]);
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    return 0;
}
