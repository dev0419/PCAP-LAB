
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA kernel to count characters in parallel
__global__ void countCharacters(char* str, int* count, int length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < length) {
        if (str[tid] != '\0') {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char hostStr[1000]; // Maximum string length is 1000 characters
    int count = 0;

    // Input: Read a string from the user
    printf("Enter a string: ");
    fgets(hostStr, sizeof(hostStr), stdin);
    hostStr[strlen(hostStr) - 1] = '\0'; // Remove the newline character

    int length = strlen(hostStr);

    // Device variables
    char* devStr;
    int* devCount;

    // Allocate memory on the GPU
    hipMalloc((void**)&devStr, length * sizeof(char));
    hipMalloc((void**)&devCount, sizeof(int));

    // Copy the string from host to device
    hipMemcpy(devStr, hostStr, length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(devCount, &count, sizeof(int), hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = 256;
    int gridSize = (length + blockSize - 1) / blockSize;

    // Launch the kernel
    countCharacters<<<gridSize, blockSize>>>(devStr, devCount, length);

    // Copy the result back from the device
    hipMemcpy(&count, devCount, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(devStr);
    hipFree(devCount);

    // Output the character count
    printf("Number of characters: %d\n", count);

    return 0;
}
