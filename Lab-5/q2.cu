#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void vecAddKernel_2(float * A, float * B, float * C, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) C[idx] = A[idx] + B[idx];
}

void vecAdd(float * A, float * B, float * C, int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;
  hipMalloc((void ** ) & d_A, size);
  hipMalloc((void ** ) & d_B, size);
  hipMalloc((void ** ) & d_C, size);
  
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  printf("A: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", A[i]);
  printf("\n");

  printf("B: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", B[i]);
  printf("\n\n");



  vecAddKernel_2<<<(n + 255) / 256, 256>>>(d_A, d_B, d_C, n);
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  printf("A+B (from 2 kernel): ");

  for (int i = 0; i < n; i++)
    printf("%f, ", C[i]);
  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() {
  float * h_A, * h_B, * h_C;
  int n = 5, size = n * sizeof(float);
  h_A = (float * ) malloc(size);
  h_B = (float * ) malloc(size);
  h_C = (float * ) malloc(size);

  for (int i = 0; i < n; i++) {
    h_A[i] = (i + 1) * 10;
    h_B[i] = i + 1;
  }
  vecAdd(h_A, h_B, h_C, n);
  return 0;
}