#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void convolutionKernel(float *N, float *M, float *P, int width, int mask_width) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < width) {
        int half_mask_width = mask_width / 2;
        float sum = 0.0;

        for (int j = 0; j < mask_width; j++) {
            int idx = tid - half_mask_width + j;
            if (idx >= 0 && idx < width) {
                sum += N[idx] * M[j];
            }
        }

        P[tid] = sum;
    }
}

int main() {
    int width = 10;
    int mask_width = 3;

    float *h_N = (float *)malloc(width * sizeof(float));
    float *h_M = (float *)malloc(mask_width * sizeof(float));
    float *h_P = (float *)malloc(width * sizeof(float));

    for (int i = 0; i < width; i++) {
        h_N[i] = i + 1;
    }

    for (int i = 0; i < mask_width; i++) {
        h_M[i] = 0.5;
    }

    float *d_N, *d_M, *d_P;
    hipMalloc((void **)&d_N, width * sizeof(float));
    hipMalloc((void **)&d_M, mask_width * sizeof(float));
    hipMalloc((void **)&d_P, width * sizeof(float));

    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (width + blockSize - 1) / blockSize;

    convolutionKernel<<<numBlocks, blockSize>>>(d_N, d_M, d_P, width, mask_width);

    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Input Array (N): ");
    for (int i = 0; i < width; i++)
        printf("%.2f, ", h_N[i]);
    printf("\n");

    printf("Mask Array (M): ");
    for (int i = 0; i < mask_width; i++)
        printf("%.2f, ", h_M[i]);
    printf("\n");

    printf("Result Array (P) after Convolution: ");
    for (int i = 0; i < width; i++)
        printf("%.2f, ", h_P[i]);
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    free(h_N);
    free(h_M);
    free(h_P);

    return 0;
}
