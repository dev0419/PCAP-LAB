#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"


__global__ void vecAddKernel_1a(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

__global__ void vecAddKernel_1b(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

__global__ void vecAddKernel_1c(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

void printArray(float* a, int n) {
    for (int i = 0; i < n; i++)
        printf("%f ", a[i]);
    printf("\n");
}

void vecAdd(float* a, float* b, float* c, int n) {
    float* d_A, *d_B, *d_C;
    int size = n * sizeof(float);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);
    printf("Array A:\n");
    printArray(a, n);
    printf("Array B:\n");
    printArray(b, n);
    
    vecAddKernel_1a<<<n, 1>>>(d_A, d_B, d_C, n);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
    printf("A + B (n blocks):\n");
    printArray(c, n);

    vecAddKernel_1b<<<1, n>>>(d_A, d_B, d_C, n);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
    printf("A + B (n threads):\n");
    printArray(c, n);

    vecAddKernel_1c<<<ceil((float)n / 256), 256>>>(d_A, d_B, d_C, n);
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);
    printf("A + B (varying block size and 256 threads):\n");
    printArray(c, n);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    float* a, *b, *c;
    int n;
    printf("Enter the size of the array:\n");
    scanf("%d", &n);
    int size = n * sizeof(float);
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);
    printf("Enter the elements in A:\n");
    for (int i = 0; i < n; i++)
        scanf("%f", &a[i]);
    printf("Enter the elements in B:\n");
    for (int i = 0; i < n; i++)
        scanf("%f", &b[i]);
    vecAdd(a, b, c, n);
    free(a); 
    free(b);
    free(c);

    return 0;
}
