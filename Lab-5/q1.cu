#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vecAddKernel_1a(float *A, float *B, float *C, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
        C[idx] = A[idx] + B[idx];
}

__global__ void vecAddKernel_1b(float *A, float *B, float *C, int n) {
    int idx = threadIdx.x;
    if (idx < n)
        C[idx] = A[idx] + B[idx];
}

int main() {
    int n = 5, size = n * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        h_A[i] = (i + 1) * 10;
        h_B[i] = i + 1;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int blockSize = n;
    int numBlocks = 1;

    vecAddKernel_1a<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("A + B (from 1a kernel): ");
    for (int i = 0; i < n; i++)
        printf("%f, ", h_C[i]);
    printf("\n");

    vecAddKernel_1b<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("A + B (from 1b kernel): ");
    for (int i = 0; i < n; i++)
        printf("%f, ", h_C[i]);
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
