#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"


__global__ void sineAngle(float* angles, float* sineVal, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        float angle = angles[idx];
        sineVal[idx] = sin(angle); // Assign the sine value to the output array
    }
}

void printArray(float* a, int n) {
    for (int i = 0; i < n; i++)
        printf("%f ", a[i]);
    printf("\n");
}

void sinAngle(float* angles, float* sineVal, int n) {
    float* d_Angle, *d_sinVal;
    int size = n * sizeof(float);
    hipMalloc((void**)&d_Angle, size);
    hipMalloc((void**)&d_sinVal, size);
    hipMemcpy(d_Angle, angles, size, hipMemcpyHostToDevice);
    printf("Angles:\n");
    printArray(angles, n);
    sineAngle<<<ceil((float)n / 256), 256>>>(d_Angle, d_sinVal, n);
    hipMemcpy(sineVal, d_sinVal, size, hipMemcpyDeviceToHost);
    printf("Sine Values are:\n");
    printArray(sineVal, n);
    hipFree(d_Angle);
    hipFree(d_sinVal);
}

int main() {
    float* angles, *sinVal;
    int n;
    printf("Enter the size of the angles array:\n");
    scanf("%d", &n);
    int size = n * sizeof(float);
    angles = (float*)malloc(size);
    sinVal = (float*)malloc(size);
    printf("Enter the angles in the array:\n");
    for (int i = 0; i < n; i++)
        scanf("%f", &angles[i]);
    sinAngle(angles, sinVal, n);
    free(angles);
    free(sinVal);
    return 0;
}
