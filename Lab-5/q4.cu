#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


// CUDA kernel to calculate sine of angles
__global__ void calculateSine(float *angles, float *sineResults, int numAngles) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numAngles) {
        sineResults[tid] = sinf(angles[tid]);
    }
}

int main() {
    int numAngles = 10; // Number of angles
    float h_angles[] = {0.0, 0.5, 1.0, 1.5, 2.0, 2.5, 3.0, 3.5, 4.0, 4.5}; // Input angles
    float h_sineResults[numAngles]; // Resultant sine values

    // Device copies of variables angles and sineResults
    float *d_angles, *d_sineResults;

    int size = numAngles * sizeof(float); // Size of the arrays in bytes

    // Allocate space for device copies of angles and sineResults
    hipMalloc((void **)&d_angles, size);
    hipMalloc((void **)&d_sineResults, size);

    // Copy input angles from host to device
    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numAngles + blockSize - 1) / blockSize;

    // Launch the CUDA kernel to calculate sine values
    calculateSine<<<numBlocks, blockSize>>>(d_angles, d_sineResults, numAngles);

    // Copy the result back to the host
    hipMemcpy(h_sineResults, d_sineResults, size, hipMemcpyDeviceToHost);

    // Display the input angles and their corresponding sine values
    printf("Input Angles (in radians):\n");
    for (int i = 0; i < numAngles; i++) {
        printf("%.2f ", h_angles[i]);
    }
    printf("\n\nSine Results:\n");
    for (int i = 0; i < numAngles; i++) {
        printf("%.4f ", h_sineResults[i]);
    }
    printf("\n");

    // Cleanup: Free device memory
    hipFree(d_angles);
    hipFree(d_sineResults);

    return 0;
}
