#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

__global__ void convolutionKernel(int* N, int* M, int* P, int width, int mask_width) {
    int pval = 0;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int start = i - (mask_width / 2);
    for (int j = 0; j < mask_width; j++) {    
        if (start+j >= 0 && start+j < width) {
            pval += N[start + j] * M[j];
        }
    }
    P[i] = pval;
}

void performConvolution(int* N, int* M, int* P, int width, int mask_width) {
    int *d_N, *d_M, *d_P;
    int size = width * sizeof(int);

    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_M, mask_width * sizeof(int));
    hipMalloc((void**)&d_P, size);

    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);

    int gridSize = (width + 255) / 256;  // Assuming 256 threads per block
    int blockSize = 256;

    convolutionKernel<<<gridSize, blockSize>>>(d_N, d_M, d_P, width, mask_width);

    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

int main() {
    int width, mask_width;
    printf("Enter the width:\n");
    scanf("%d", &width);
    printf("Enter the mask width of the array:\n");
    scanf("%d", &mask_width);

    int* N = (int*)malloc(sizeof(int) * width);
    int* M = (int*)malloc(sizeof(int) * mask_width);
    int* P = (int*)malloc(sizeof(int) * width);

    printf("Enter the elements in the array:\n");
    for (int i = 0; i < width; i++)
        scanf("%d", &N[i]);
    printf("Enter the elements in the mask:\n");
    for (int i = 0; i < mask_width; i++)
        scanf("%d", &M[i]);

    performConvolution(N, M, P, width, mask_width);

    printf("Result:\n");
    for (int i = 0; i < width; i++) {
        printf("%d ", P[i]);
    }

    free(N);
    free(M);
    free(P);

    return 0;
}
