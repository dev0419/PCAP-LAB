#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define MASK_WIDTH 3
#define WIDTH 8

__global__ void conv1D(int* input, int* mask, int* output, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int half_mask = MASK_WIDTH / 2;
    int value = 0;

    for (int i = -half_mask; i <= half_mask; i++) {
        int idx_shifted = idx + i;
        if (idx_shifted >= 0 && idx_shifted < width) {
            value += input[idx_shifted] * mask[i + half_mask];
        }
    }

    output[idx] = value;
}

int main() {
    int* input, * mask, * output, * d_input, * d_mask, * d_output;
    int input_size = WIDTH * sizeof(int);
    int mask_size = MASK_WIDTH * sizeof(int);

    // Allocate and initialize input and mask arrays
    input = (int*)malloc(input_size);
    mask = (int*)malloc(mask_size);

    printf("Enter %d values for the input array:\n", WIDTH);
    for (int i = 0; i < WIDTH; i++) {
        scanf("%d", &input[i]);
    }

    printf("Enter %d values for the mask:\n", MASK_WIDTH);
    for (int i = 0; i < MASK_WIDTH; i++) {
        scanf("%d", &mask[i]);
    }

    output = (int*)malloc(input_size);

    // Allocate device memory
    hipMalloc((void**)&d_input, input_size);
    hipMalloc((void**)&d_mask, mask_size);
    hipMalloc((void**)&d_output, input_size);

    // Copy data from host to device
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_size, hipMemcpyHostToDevice);

    // Launch the convolution kernel
    int num_blocks = (WIDTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    conv1D<<<num_blocks, THREADS_PER_BLOCK>>>(d_input, d_mask, d_output, WIDTH);

    // Copy the result back to the host
    hipMemcpy(output, d_output, input_size, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result:\n");
    for (int i = 0; i < WIDTH; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
    free(input);
    free(mask);
    free(output);

    return 0;
}
