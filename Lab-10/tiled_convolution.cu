#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void convolution(int* N, int* M, int* P, int width, int mask_width) {
    int tid = threadIdx.x;
    int start = tid - mask_width/2;
    int pval = 0;

    for (int i = 0; i < mask_width; i++) {
        if (start + i >= 0 && start + i < width) {
            pval += N[start + i] * M[i];
        }
    }

    P[tid] = pval;
    __syncthreads();
}

void performConvolution(int* N, int* M, int* P,int width,int mask_width) {
    int* d_N, *d_M, *d_P;
    int size = width * sizeof(int);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_M, mask_width * sizeof(int));  
    hipMalloc((void**)&d_P, size);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int),hipMemcpyHostToDevice);  
    convolution<<<1,width>>>(d_N, d_M, d_P, width,mask_width);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

int main() {
    int width,mask_width;
    printf("Enter the width:\n");
    scanf("%d", &width);
    printf("Enter the mask width:\n");
    scanf("%d",&mask_width);
    int* N = (int*)malloc(sizeof(int) * width);
    int* M = (int*)malloc(sizeof(int) * mask_width);  
    int* P = (int*)malloc(sizeof(int) * width);
    printf("Enter the elements in the array:\n");
    for (int i = 0; i < width; i++)
        scanf("%d", &N[i]);
    printf("Enter the elements in the mask:\n");
    for (int i = 0; i < mask_width; i++)  
        scanf("%d", &M[i]);
    performConvolution(N, M, P, width,mask_width);
    printf("Result:\n");
    for (int i = 0; i < width; i++)
        printf("%d ", P[i]);
    free(M);
    free(N);
    free(P);
    return 0;
}


