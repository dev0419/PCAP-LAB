#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define TILE_SIZE 32
#define MASK_WIDTH 5  

__global__ void convolution(int* N, int* M, int* P, int width) {
    __shared__ int N_tile[TILE_SIZE + 2 * (MASK_WIDTH / 2)];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int start = i - (MASK_WIDTH / 2);
    int pval = 0;
    int local_index = threadIdx.x + MASK_WIDTH / 2;

    if (start >= 0 && start < width) {
        N_tile[local_index] = N[start];
    } else {
        N_tile[local_index] = 0;
    }
    __syncthreads();

    for (int j = 0; j < MASK_WIDTH; j++) {
        pval += N_tile[local_index + j - MASK_WIDTH / 2] * M[j];
    }

    P[i] = pval;
}

void performConvolution(int* N, int* M, int* P, int width) {
    int* d_N, *d_M, *d_P;
    int size = width * sizeof(int);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_M, MASK_WIDTH * sizeof(int));  
    hipMalloc((void**)&d_P, size);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, MASK_WIDTH * sizeof(int),hipMemcpyHostToDevice);  
    int gridSize = (width + TILE_SIZE - 1) / TILE_SIZE;
    int blockSize = TILE_SIZE;
    convolution<<<gridSize, blockSize>>>(d_N, d_M, d_P, width);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

int main() {
    int width;
    printf("Enter the width:\n");
    scanf("%d", &width);
    int* N = (int*)malloc(sizeof(int) * width);
    int* M = (int*)malloc(sizeof(int) * MASK_WIDTH);  
    int* P = (int*)malloc(sizeof(int) * width);
    printf("Enter the elements in the array:\n");
    for (int i = 0; i < width; i++)
        scanf("%d", &N[i]);
    printf("Enter the elements in the mask:\n");
    for (int i = 0; i < MASK_WIDTH; i++)  
        scanf("%d", &M[i]);
    performConvolution(N, M, P, width);
    printf("Result:\n");
    for (int i = 0; i < width; i++)
        printf("%d ", P[i]);
    free(M);
    free(N);
    free(P);
}
