#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void convolution(int* N,int* M,int* P,int width,int mask_width){
    __shared__ int value;
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int start = tid - (mask_width/2);
    P[tid] = 0;
    for(int i = 0;i < mask_width;i++){
        if(tid == 0)
            value = M[i];
        __syncthreads();
        if(start + i >= 0 && start + i < width)
            P[tid] += N[start+i]*value;  
        __syncthreads();    
    }
}

void performConvolution(int* N,int* M,int* P,int width,int mask_width){
    int* d_N,*d_M,*d_P;
    int size = width*sizeof(int), mask_size = mask_width*sizeof(int);
    hipMalloc((void**)&d_N,size);
    hipMalloc((void**)&d_M,mask_size);
    hipMalloc((void**)&d_P,size);
    hipMemcpy(d_N,N,size,hipMemcpyHostToDevice);
    hipMemcpy(d_M,M,mask_size,hipMemcpyHostToDevice);
    convolution<<<1,width>>>(d_N,d_M,d_P,width,mask_width);
    hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);
    hipFree(d_P);
    hipFree(d_M);
    hipFree(d_N);
}

int main(){
    int* N,*M,*P,width,mask_width;
    printf("Enter the width:\n");
    scanf("%d",&width);
    printf("Enter the mask width:\n");
    scanf("%d",&mask_width);
    int size = width*sizeof(int), mask_size = mask_width*sizeof(int);
    N = (int*)malloc(size);
    M = (int*)malloc(mask_size);
    P = (int*)malloc(size);
    printf("Enter array elements:\n");
    for(int i = 0;i < width;i++)
        scanf("%d",&N[i]);
    printf("Enter the mask elements:\n");
    for(int i = 0;i < mask_width;i++)
        scanf("%d",&M[i]);
    performConvolution(N,M,P,width,mask_width);
    printf("Result:\n");
    for(int i = 0;i < width;i++)
        printf("%d ",P[i]);
    free(P);
    free(M);
    free(N);
    return 0;
}
