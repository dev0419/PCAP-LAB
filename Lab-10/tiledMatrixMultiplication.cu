#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void matMul(int* a,int* b,int* c){
    __shared__ int MD[TILE_WIDTH][TILE_WIDTH];
    __shared__  int ND[TILE_WIDTH][TILE_WIDTH];
    int m;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;
    int pval = 0;
    for(m = 0;m < WIDTH/TILE_WIDTH;m++){
        MD[tx][ty] = a[row*WIDTH + m * TILE_WIDTH+tx];
        ND[tx][ty] = b[(m*TILE_WIDTH + ty)* WIDTH+col];
        __syncthreads();
        for(int k = 0;k < TILE_WIDTH;k++){
            pval += MD[ty][k]*ND[k][tx];
        }

        __syncthreads();
    }
    c[row*WIDTH + col] = pval;
}


int main(){
    int* matA,*matB,*matC,*d_a,*d_b,*d_c;
    matA = (int*)malloc(WIDTH*WIDTH*sizeof(int));
    printf("Enter the elements of 4x4 matA:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
      scanf("%d",&matA[i]);
    matB = (int*)malloc(WIDTH*WIDTH*sizeof(int));
    printf("Enter the elements of 4x4 matB:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
      scanf("%d",&matB[i]);
    matC = (int*)malloc(WIDTH*WIDTH*sizeof(int));
    hipMalloc((void**)&d_a,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&d_b,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&d_c,sizeof(int)*WIDTH*WIDTH);
    hipMemcpy(d_a,matA,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(d_b,matB,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH,TILE_WIDTH);
    matMul<<<grid_conf,block_conf>>>(d_a,d_b,d_c);
    hipMemcpy(matC,d_c,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < WIDTH;i++){
      for(int j = 0;j < WIDTH;j++){ 
        printf("%d ",matC[i*WIDTH + j]);
      }
      printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(matA);
    free(matB);
    free(matC);
    return 0;
}
