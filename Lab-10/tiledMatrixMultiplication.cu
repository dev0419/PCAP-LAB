#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void matMul(int* a,int* b,int* c){
    __shared__ int MD[TILE_WIDTH][TILE_WIDTH];
    __shared__ int ND[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;
    int pval = 0;
    int m;
    for(m = 0;m < WIDTH/TILE_WIDTH;m++){
        MD[tx][ty] = a[row*WIDTH + m*TILE_WIDTH + tx];
        ND[tx][ty] = b[m*TILE_WIDTH+ ty*WIDTH + col];
        __syncthreads();
    }
    for(int k = 0;k < TILE_WIDTH;k++){
        pval += MD[ty][k] * ND[k][tx];
    }
    __syncthreads();
    c[row*WIDTH + col] = pval;
}

int main(){
    int* a,*b,*c,*da,*db,*dc;
    a = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    b = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    c = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    printf("Enter the 4x4 matrix a:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&a[i]);
    printf("Enter the 4x4 matrix b:\n");
    for(int i = 0;i < WIDTH*WIDTH;i++)
        scanf("%d",&b[i]);
    hipMalloc((void**)&da,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&db,sizeof(int)*WIDTH*WIDTH);
    hipMalloc((void**)&dc,sizeof(int)*WIDTH*WIDTH);
    hipMemcpy(da,a,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(db,b,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    dim3 grid_conf(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH);
    dim3 block_conf(TILE_WIDTH,TILE_WIDTH);
    matMul<<<grid_conf,block_conf>>>(da,db,dc);
    hipMemcpy(c,dc,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < WIDTH;i++){
        for(int j = 0;j < WIDTH;j++){
            printf("%d ",c[i*WIDTH + j]);
        }
        printf("\n");
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    return 0;
}
