#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__device__ int co_rank(int k,int* a,int m,int* b,int n){
    int i = min(k,m);
    int j = k - i;
    int i_low = max(0,k - n);
    int j_low = max(0,k - m);
    bool flag = true;
    int delta;
    while(flag){
        if(i > 0 && j < n && a[i - 1] > b[j]){
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            i -=  delta;
            j += delta;
        } else if(i < m && j > 0 && b[j - 1] >= a[i]){
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            j -= delta;
            i += delta;
        }
        else{
            flag = false;
        }
    }
    return i;
}

__device__ void merge_sequential(int* a,int m,int* b,int n,int* c){
    int i = 0;
    int j = 0;
    int k = 0;
    while(i < m && j < n){
        if(a[i] < b[j]){
            c[k++] = a[i++]; 
        } else if(b[j] < a[i]){
            c[k++] = b[j++];
        }
    } while(i < m){
        c[k++] = a[i++];
    } while(j < n){
        c[k++] = b[j++];
    }
}

__global__ void merge(int* a,int m,int* b,int n,int* c){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int total = m + n;
    int elementsPerThread = ceil((double)total/(gridDim.x*blockDim.x));
    int k_curr = tid*elementsPerThread;
    int k_next = min((int)(tid + 1)*elementsPerThread,total);
    int i_curr = co_rank(k_curr,a,m,b,n);
    int i_next = co_rank(k_next,a,m,b,n);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;
    if(tid < total){
        merge_sequential(a + i_curr,i_next - i_curr,b + j_curr,j_next - j_curr,c + k_curr);
    }
}

int main(){
    int m,n,*a,*b,*c,*da,*db,*dc;
    printf("Enter size m,n:\n");
    scanf("%d %d",&m,&n);
    a = (int*)malloc(sizeof(int)*m);
    b = (int*)malloc(sizeof(int)*n);
    c = (int*)malloc(sizeof(int)*(m+n));
    printf("Enter the sorted array a:\n");
    for(int i = 0;i < m;i++)
        scanf("%d",&a[i]);
    printf("Enter the sorted array b:\n");
    for(int i = 0;i < n;i++)
        scanf("%d",&b[i]);
    hipMalloc((void**)&da,sizeof(int)*m);
    hipMalloc((void**)&db,sizeof(int)*n);
    hipMalloc((void**)&dc,sizeof(int)*(m+n));
    hipMemcpy(da,a,sizeof(int)*m,hipMemcpyHostToDevice);
    hipMemcpy(db,b,sizeof(int)*n,hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = (int)ceil((double)(m + n)/blockSize);
    merge<<<gridSize,blockSize>>>(da,m,db,n,dc);
    hipMemcpy(c,dc,sizeof(int)*(m+n),hipMemcpyDeviceToHost);
    printf("Resulting array:\n");
    for(int i = 0;i < (m+n);i++)
        printf("%d ",c[i]);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}
