#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ int co_rank(int k, int* a, int m, int* b, int n){
    int i = min(k, m);
    int j = k - i;
    int i_low = max(0, k - n);
    int j_low = max(0, k - m);
    int delta;
    bool flag = true;
    while(flag){
        if(i > 0 && j < n && a[i - 1] > b[j]){
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            i -= delta;
            j += delta;
        } else if(j > 0 && i < m && b[j - 1] >= a[i]){
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i += delta;
            j -= delta; 
        } else{
            flag = false;
        }
    }
    return i;
}

__device__ void merge_sequential(int* a, int a_count, int* b, int b_count, int* c){
    int i = 0, j = 0, k = 0;
    while(i < a_count && j < b_count){
        c[k++] = (a[i] <= b[j]) ? a[i++] : b[j++];
    }
    while(i < a_count){
        c[k++] = a[i++];
    }
    while(j < b_count){
        c[k++] = b[j++];
    }
}

__global__ void merge_kernel(int* a, int m, int* b, int n, int* c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = m + n;
    int k_curr = tid * ceilf((float)total / (blockDim.x * gridDim.x));
    int k_next = min((int)((tid + 1) * ceilf((float)total / (blockDim.x * gridDim.x))), total);
    int i_curr = co_rank(k_curr, a, m, b, n);
    int i_next = co_rank(k_next, a, m, b, n);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;
    if(tid < total) {
        merge_sequential(a + i_curr, i_next - i_curr, b + j_curr, j_next - j_curr, c + k_curr);
    }
}

int main(){
    int m, n;
    int *a, *b, *c;
    int *da, *db, *dc;
    printf("Enter the size of array A (m) and array B (n):\n");
    scanf("%d %d", &m, &n);
    a = (int*)malloc(m * sizeof(int));
    b = (int*)malloc(n * sizeof(int));
    c = (int*)malloc((m + n) * sizeof(int));
    printf("Enter the sorted elements for array A:\n");
    for(int i = 0; i < m; i++)
        scanf("%d", &a[i]);
    printf("Enter the sorted elements for array B:\n");
    for(int i = 0; i < n; i++)
        scanf("%d", &b[i]);
  
    hipMalloc((void**)&da, m * sizeof(int));
    hipMalloc((void**)&db, n * sizeof(int));
    hipMalloc((void**)&dc, (m + n) * sizeof(int));
    hipMemcpy(da, a, m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = (int)ceil((float)(m + n) / blockSize);
    merge_kernel<<<gridSize, blockSize>>>(da, m, db, n, dc);
    hipMemcpy(c, dc, (m + n) * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resulting merged array:\n");
    for(int i = 0; i < (m + n); i++)
        printf("%d ", c[i]);
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    return 0;
}
