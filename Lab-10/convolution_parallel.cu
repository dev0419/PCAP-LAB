#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define MASK_SIZE 3

__constant__ int mask[MASK_SIZE];

__global__ void convolution(int* N,int* P,int width,int mask_width){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < width){
        int res = 0;
        int start = tid  - mask_width/2;
        for(int i = 0;i < mask_width;i++){
            if(start + i >= 0 && start + i < width){
                res += N[start + i]*mask[i];
            }
        }
        P[tid] = res;
    }
}

void performConvolution(int* N,int* M,int* P,int width,int mask_width){
    int* d_N,*d_P;
    int size = sizeof(int)*width;
    int mask_size = sizeof(int)*mask_width;
    hipMalloc((void**)&d_N,size);
    hipMalloc((void**)&d_P,size);
    hipMemcpy(d_N,N,size,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask),M,mask_size);
    int blockSize = 256;
    int gridSize = (width + blockSize - 1)/ blockSize;
    convolution<<<gridSize,blockSize>>>(d_N,d_P,width,mask_width);
    hipMemcpy(P,d_P,size,hipMemcpyDeviceToHost);
    hipFree(d_N);
    hipFree(d_P);
}

int main(){
    int* N,*M,*P,width,mask_width;
    printf("Enter the width:\n");
    scanf("%d",&width);
    printf("Enter the mask width:\n");
    scanf("%d",&mask_width);
    int size = sizeof(int)*width;
    int mask_size = sizeof(int)*mask_width;
    N = (int*)malloc(size);
    M = (int*)malloc(mask_size);
    P = (int*)malloc(size);
    printf("Enter the array elements:\n");
    for(int i = 0;i < width;i++)
        scanf("%d",&N[i]);
    printf("Enter the mask elements:\n");
    for(int i = 0; i < mask_width;i++)
        scanf("%d",&M[i]);
    performConvolution(N,M,P,width,mask_width);
    printf("Result:\n");
    for(int i = 0;i < width;i++)
        printf("%d ", P[i]);
    free(N);
    free(M);
    free(P);
    return 0;
}
