#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define MASK_SIZE 3
__constant__ int mask[MASK_SIZE];

__global__ void convolution(int* n,int width,int* p,int mask_width){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int start = tid - (mask_width/2);
    int res = 0;
    if(tid < width){
        for(int i = 0;i < mask_width;i++){
            if(start + i < width){
                res += n[start + i]*mask[i];
            }
        }
        p[tid] = res;
    }
}

int main(){
    int width,mask_width,*n,*m,*p,*dn,*dp;
    printf("Enter the width:\n");
    scanf("%d",&width);
    printf("Enter the mask width:\n");
    scanf("%d",&mask_width);
    n = (int*)malloc(sizeof(int)*width);
    m = (int*)malloc(sizeof(int)*mask_width);
    p = (int*)malloc(sizeof(int)*width);
    printf("Enter the array n:\n");
    for(int i = 0;i < width;i++)
        scanf("%d",&n[i]);
    printf("Enter the mask:\n");
    for(int i = 0;i < mask_width;i++)
        scanf("%d",&m[i]);
    hipMalloc((void**)&dn,sizeof(int)*width);
    hipMalloc((void**)&dp,sizeof(int)*width);
    hipMemcpy(dn,n,sizeof(int)*width,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask),m,sizeof(int)*mask_width);
    int blockSize = 256;
    int gridSize = (width + blockSize - 1)/blockSize;
    convolution<<<gridSize,blockSize>>>(dn,width,dp,mask_width);
    hipMemcpy(p,dp,sizeof(int)*width,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < width;i++)
        printf("%d ",p[i]);
    hipFree(dn);
    hipFree(dp);
    free(n);
    free(p);
    return 0;
}
