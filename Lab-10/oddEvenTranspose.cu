#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void odd(int* a, int n){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid % 2 != 0 && tid + 1 < n){
        if(a[tid] > a[tid + 1]){
            int temp = a[tid];
            a[tid] = a[tid+1];
            a[tid + 1] = temp;
        }
    }
}

__global__ void even(int* a, int n){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid % 2 == 0 && tid + 1 < n){
        if(a[tid] > a[tid + 1]){
            int temp = a[tid];
            a[tid] = a[tid+1];
            a[tid + 1] = temp;
        }
    }
}

int main(){
    int* a,*da,n;
    printf("Enter n:\n");
    scanf("%d",&n);
    a = (int*)malloc(sizeof(int)*n);
    printf("Enter the array elements:\n");
    for(int i = 0;i < n;i++)
        scanf("%d",&a[i]);
    hipMalloc((void**)&da,n*sizeof(int));
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    for(int i = 0; i < n/2;i++){
        odd<<<1,n>>>(da,n);
        even<<<1,n>>>(da,n);
    }
    hipMemcpy(a,da,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < n;i++)
        printf("%d ",a[i]);
    hipFree(da);
    free(a);
    return 0;
}
