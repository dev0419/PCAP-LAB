#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void odd(int* a,int n){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid % 2 != 0 && tid + 1 < n){
        if(a[tid] > a[tid + 1]){
            int temp = a[tid];
            a[tid] = a[tid + 1];
            a[tid + 1] = temp;
        }
    }
}

__global__ void even(int* a,int n){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid % 2 == 0 && tid + 1 < n){
        if(a[tid] > a[tid + 1]){
            int temp = a[tid];
            a[tid] = a[tid + 1];
            a[tid + 1] = temp;
        }
    }
}

int main(){
    int* a,n,*da;
    printf("Enter n:\n");
    scanf("%d", &n);
    printf("Enter the array elements:\n");
    a = (int*)malloc(sizeof(int)*n);
    for(int i = 0;i < n;i++){
        scanf("%d",&a[i]);
    }
    hipMalloc((void**)&da,sizeof(int)*n);
    hipMemcpy(da,a,sizeof(int)*n,hipMemcpyHostToDevice);
    for(int i = 0;i < n;i++){
        odd<<<1,n>>>(da,n);
        even<<<1,n>>>(da,n);
    }
    hipMemcpy(a,da,sizeof(int)*n,hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(int i = 0;i < n;i++){
        printf("%d ",a[i]);
    }
    hipFree(da);
    free(a);
    return 0;
}
