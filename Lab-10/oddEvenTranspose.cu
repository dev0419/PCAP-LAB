#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void odd(int *a, int n) {
    int tid = threadIdx.x * 2;
    if (tid + 1 < n) {
        if (a[tid] > a[tid + 1]) {
            int temp = a[tid];
            a[tid] = a[tid + 1];
            a[tid + 1] = temp;
        }
    }
}

__global__ void even(int *a, int n) {
    int tid = threadIdx.x * 2 + 1;
    if (tid + 1 < n) {
        if (a[tid] > a[tid + 1]) {
            int temp = a[tid];
            a[tid] = a[tid + 1];
            a[tid + 1] = temp;
        }
    }
}

int main() {
    int *a, n, *d_a;
    printf("Enter the size of the array:\n");
    scanf("%d", &n);
    printf("Enter the array elements:\n");
    a = (int*)malloc(sizeof(int) * n);
    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }
    hipMalloc((void**)&d_a, sizeof(int) * n);
    hipMemcpy(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice);
    for (int i = 0; i < n / 2; i++) {
        odd<<<1, n / 2>>>(d_a, n);
        even<<<1, n / 2>>>(d_a, n);
    }
    hipMemcpy(a, d_a, sizeof(int) * n, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < n; i++)
        printf("%d ", a[i]);
    hipFree(d_a);
    free(a);
    return 0;
}
